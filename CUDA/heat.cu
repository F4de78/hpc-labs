
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

// Simple define to index into a 1D array from 2D space
#define I2D(row_len, c, r) ((r) * (row_len) + (c))

__global__ void step_kernel_mod(int ni, int nj, float fact, float *temp_in, float *temp_out) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int j = blockIdx.y * blockDim.y + threadIdx.y;

  if (i >= 1 && i < ni - 1 && j >= 1 && j < nj - 1) {
    int i00 = I2D(ni, i, j);
    int im10 = I2D(ni, i - 1, j);
    int ip10 = I2D(ni, i + 1, j);
    int i0m1 = I2D(ni, i, j - 1);
    int i0p1 = I2D(ni, i, j + 1);

    float d2tdx2 = temp_in[im10] - 2 * temp_in[i00] + temp_in[ip10];
    float d2tdy2 = temp_in[i0m1] - 2 * temp_in[i00] + temp_in[i0p1];

    temp_out[i00] = temp_in[i00] + fact * (d2tdx2 + d2tdy2);
  }
}

void step_kernel_ref(int ni, int nj, float fact, float *temp_in, float *temp_out)
{
  int i00, im10, ip10, i0m1, i0p1;
  float d2tdx2, d2tdy2;
  //printf("%d %d\n", ni, nj);
  // loop over all points in domain (except boundary)
  for (int j = 1; j < nj - 1; j++)
  {
    for (int i = 1; i < ni - 1; i++)
    {
      // find indices into linear memory
      // for central point and neighbours
      i00 = I2D(ni, i, j);
      im10 = I2D(ni, i - 1, j);
      ip10 = I2D(ni, i + 1, j);
      i0m1 = I2D(ni, i, j - 1);
      i0p1 = I2D(ni, i, j + 1);

      // evaluate derivatives
      d2tdx2 = temp_in[im10] - 2 * temp_in[i00] + temp_in[ip10];
      d2tdy2 = temp_in[i0m1] - 2 * temp_in[i00] + temp_in[i0p1];

      // update temperatures
      temp_out[i00] = temp_in[i00] + fact * (d2tdx2 + d2tdy2);
    }
  }
}

int main()
{
  int istep;
  int nstep = 200; // number of time steps

  // Specify our 2D dimensions
  const int ni = 1000;
  const int nj = 1000;
  float tfac = 8.418e-5; // thermal diffusivity of silver

  float *temp1_ref, *temp2_ref, *temp1, *temp2, *temp_tmp, *cpu_arr;

  const int size = ni * nj * sizeof(float);

  temp1_ref = (float *)malloc(size);
  temp2_ref = (float *)malloc(size);
  cpu_arr = (float *)malloc(size);

  hipMalloc((void **)&temp1, size);
  hipMalloc((void **)&temp2, size);

  // Initialize with random data
  for (int i = 0; i < ni * nj; ++i)
  {
    float rnd = (float)rand() / (float)(RAND_MAX / 100.0f);
    temp1_ref[i] = temp2_ref[i] = rnd;
  }

  hipMemcpy(temp1, temp1_ref, size, hipMemcpyHostToDevice);
  hipMemcpy(temp2, temp2_ref, size, hipMemcpyHostToDevice);

  // Execute the CPU-only reference version
  for (istep = 0; istep < nstep; istep++)
  {
    step_kernel_ref(ni, nj, tfac, temp1_ref, temp2_ref);

    // swap the temperature pointers
    temp_tmp = temp1_ref;
    temp1_ref = temp2_ref;
    temp2_ref = temp_tmp;
  }

  // Execute the modified version using same data
  // https://developer.nvidia.com/blog/cuda-refresher-cuda-programming-model/
  dim3 threadsPerBlock(16, 16); // 1024 threads per block
  dim3 numBlocks((ni + threadsPerBlock.x - 1) / threadsPerBlock.x, (nj + threadsPerBlock.y - 1) / threadsPerBlock.y);
  
  for (istep = 0; istep < nstep; istep++)
  {
    step_kernel_mod<<<numBlocks, threadsPerBlock>>>(ni, nj, tfac, temp1, temp2);

    // swap the temperature pointers
    temp_tmp = temp1;
    temp1 = temp2;
    temp2 = temp_tmp;
  }

  hipMemcpy(cpu_arr, temp1, size, hipMemcpyDeviceToHost);

  float maxError = 0;
  // Output should always be stored in the temp1 and temp1_ref at this point
  for (int i = 0; i < ni * nj; ++i)
  {
    if (abs(cpu_arr[i] - temp1_ref[i]) > maxError)
    { 
      printf("cpu_arr: %f - temp1_ref:%f\n",cpu_arr[i] ,temp1_ref[i] );
      maxError = abs(cpu_arr[i] - temp1_ref[i]);
    }
  }

  // Check and see if our maxError is greater than an error bound
  if (maxError > 0.0005f)
  {
    printf("Problem! The Max Error of %.5f is NOT within acceptable bounds.\n", maxError);
  }
  else
  {
    printf("The Max Error of %.5f is within acceptable bounds.\n", maxError);
  }

  free(temp1_ref);
  free(temp2_ref);
  hipFree(temp1);
  hipFree(temp2);

  return 0;
}
