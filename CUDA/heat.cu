
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

// Simple define to index into a 1D array from 2D space
#define I2D(row_len, c, r) ((r) * (row_len) + (c))

#define BLOCKS 256
#define THREADS 512

__global__ void step_kernel_mod(int ni, int nj, float fact, float *temp_in, float *temp_out)
{
  int i00, im10, ip10, i0m1, i0p1;
  float d2tdx2, d2tdy2;

  int j = threadIdx.x + blockDim.x * blockIdx.x;
  int i = threadIdx.y + blockDim.y * blockIdx.y;

  if(i>=ni-1 || j>=nj-1){
    printf("altro bound %d %d\n",i,j );
    return;
  }
  if(i==0 || j==0){
    printf("bound %d %d\n",i,j );
    return;
  }

  printf("i:%d j:%d\n", i, j);
  // find indices into linear memory
  // for central point and neighbours
  i00 = I2D(ni, i, j);
  im10 = I2D(ni, i - 1, j);
  ip10 = I2D(ni, i + 1, j);
  i0m1 = I2D(ni, i, j - 1);
  i0p1 = I2D(ni, i, j + 1);

  // evaluate derivatives
  d2tdx2 = temp_in[im10] - 2 * temp_in[i00] + temp_in[ip10];
  d2tdy2 = temp_in[i0m1] - 2 * temp_in[i00] + temp_in[i0p1];

  // update temperatures
  temp_out[i00] = temp_in[i00] + fact * (d2tdx2 + d2tdy2);


  //  int i00, im10, ip10, i0m1, i0p1;
  // float d2tdx2, d2tdy2;

  // // loop over all points in domain (except boundary)
  // for (int j = 1; j < nj - 1; j++)
  // {
  //   for (int i = 1; i < ni - 1; i++)
  //   {
  //     // find indices into linear memory
  //     // for central point and neighbours
  //     i00 = I2D(ni, i, j);
  //     im10 = I2D(ni, i - 1, j);
  //     ip10 = I2D(ni, i + 1, j);
  //     i0m1 = I2D(ni, i, j - 1);
  //     i0p1 = I2D(ni, i, j + 1);

  //     // evaluate derivatives
  //     d2tdx2 = temp_in[im10] - 2 * temp_in[i00] + temp_in[ip10];
  //     d2tdy2 = temp_in[i0m1] - 2 * temp_in[i00] + temp_in[i0p1];

  //     // update temperatures
  //     temp_out[i00] = temp_in[i00] + fact * (d2tdx2 + d2tdy2);
  //   }
  // }
}

void step_kernel_ref(int ni, int nj, float fact, float *temp_in, float *temp_out)
{
  int i00, im10, ip10, i0m1, i0p1;
  float d2tdx2, d2tdy2;
  printf("%d %d\n", ni, nj);
  // loop over all points in domain (except boundary)
  for (int j = 1; j < nj - 1; j++)
  {
    for (int i = 1; i < ni - 1; i++)
    {
      // find indices into linear memory
      // for central point and neighbours
      i00 = I2D(ni, i, j);
      im10 = I2D(ni, i - 1, j);
      ip10 = I2D(ni, i + 1, j);
      i0m1 = I2D(ni, i, j - 1);
      i0p1 = I2D(ni, i, j + 1);

      // evaluate derivatives
      d2tdx2 = temp_in[im10] - 2 * temp_in[i00] + temp_in[ip10];
      d2tdy2 = temp_in[i0m1] - 2 * temp_in[i00] + temp_in[i0p1];

      // update temperatures
      temp_out[i00] = temp_in[i00] + fact * (d2tdx2 + d2tdy2);
    }
  }
}

int main()
{
  int istep;
  int nstep = 200; // number of time steps

  // Specify our 2D dimensions
  const int ni = 10;
  const int nj = 10;
  float tfac = 8.418e-5; // thermal diffusivity of silver

  float *temp1_ref, *temp2_ref, *temp1, *temp2, *temp_tmp, *cpu_arr;

  const int size = ni * nj * sizeof(float);

  temp1_ref = (float *)malloc(size);
  temp2_ref = (float *)malloc(size);
  cpu_arr = (float *)malloc(size);

  hipMalloc((void **)&temp1, size);
  hipMalloc((void **)&temp2, size);

  // Initialize with random data
  for (int i = 0; i < ni * nj; ++i)
  {
    float rnd = (float)rand() / (float)(RAND_MAX / 100.0f);
    temp1_ref[i] = temp2_ref[i] = rnd;
  }

  hipMemcpy(temp1, temp1_ref, size, hipMemcpyHostToDevice);
  hipMemcpy(temp2, temp2_ref, size, hipMemcpyHostToDevice);

//  printf("Temp1: 69420, Temp1ref: %f\n", temp1_ref[0]);

//  cudaMemcpy(cpu_arr, temp1, size, cudaMemcpyDeviceToHost);
//  printf("YAYAYAY %f %f\n", cpu_arr[0], cpu_arr[1]);

  // Execute the CPU-only reference version
  for (istep = 0; istep < nstep; istep++)
  {
    step_kernel_ref(ni, nj, tfac, temp1_ref, temp2_ref);

    // swap the temperature pointers
    temp_tmp = temp1_ref;
    temp1_ref = temp2_ref;
    temp2_ref = temp_tmp;
  }

  // Execute the modified version using same data
  for (istep = 0; istep < nstep; istep++)
  {
    step_kernel_mod<<<BLOCKS, THREADS>>>(ni, nj, tfac, temp1, temp2);

    // swap the temperature pointers
    // cudaMemcpy(temp1, temp2, size, cudaMemcpyDeviceToHost);
    temp_tmp = temp1;
    temp1 = temp2;
    temp2 = temp_tmp;
    //cudaMemcpy(temp1, temp2, size, cudaMemcpyDeviceToDevice);
  }
  //cudaDeviceSynchronize();

  hipMemcpy(cpu_arr, temp2, size, hipMemcpyDeviceToHost);

  float maxError = 0;
  // Output should always be stored in the temp1 and temp1_ref at this point
  for (int i = 0; i < ni * nj; ++i)
  {
    // printf("Doing stuff at cell: %d\n", i);
    //printf("era questo:%f %f\n", cpu_arr[i], temp1_ref[i]);
    if (abs(cpu_arr[i] - temp1_ref[i]) > maxError)
    {
      maxError = abs(cpu_arr[i] - temp1_ref[i]);
    }
  }

  // Check and see if our maxError is greater than an error bound
  if (maxError > 0.0005f)
  {
    printf("Problem! The Max Error of %.5f is NOT within acceptable bounds.\n", maxError);
  }
  else
  {
    printf("The Max Error of %.5f is within acceptable bounds.\n", maxError);
  }

  free(temp1_ref);
  free(temp2_ref);
  hipFree(temp1);
  hipFree(temp2);

  return 0;
}
