
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

// Simple define to index into a 1D array from 2D space
#define I2D(row_len, c, r) ((r) * (row_len) + (c))

#ifndef NI
#define NI 1000
#endif
#ifndef NJ
#define NJ 1000
#endif
#ifndef THREADS_X
#define THREADS_X 16
#endif
#ifndef THREADS_Y
#define THREADS_Y 16
#endif

__global__ void step_kernel_mod(int ni, int nj, float fact, float *temp_in, float *temp_out)
{
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int j = blockIdx.y * blockDim.y + threadIdx.y;

  if (i >= 1 && i < ni - 1 && j >= 1 && j < nj - 1)
  {
    int i00 = I2D(ni, i, j);
    int im10 = I2D(ni, i - 1, j);
    int ip10 = I2D(ni, i + 1, j);
    int i0m1 = I2D(ni, i, j - 1);
    int i0p1 = I2D(ni, i, j + 1);

    float d2tdx2 = temp_in[im10] - 2 * temp_in[i00] + temp_in[ip10];
    float d2tdy2 = temp_in[i0m1] - 2 * temp_in[i00] + temp_in[i0p1];

    temp_out[i00] = temp_in[i00] + fact * (d2tdx2 + d2tdy2);
  }
}

void step_kernel_ref(int ni, int nj, float fact, float *temp_in, float *temp_out)
{
  int i00, im10, ip10, i0m1, i0p1;
  float d2tdx2, d2tdy2;
  // printf("%d %d\n", ni, nj);
  //  loop over all points in domain (except boundary)
  for (int j = 1; j < nj - 1; j++)
  {
    for (int i = 1; i < ni - 1; i++)
    {
      // find indices into linear memory
      // for central point and neighbours
      i00 = I2D(ni, i, j);
      im10 = I2D(ni, i - 1, j);
      ip10 = I2D(ni, i + 1, j);
      i0m1 = I2D(ni, i, j - 1);
      i0p1 = I2D(ni, i, j + 1);

      // evaluate derivatives
      d2tdx2 = temp_in[im10] - 2 * temp_in[i00] + temp_in[ip10];
      d2tdy2 = temp_in[i0m1] - 2 * temp_in[i00] + temp_in[i0p1];

      // update temperatures
      temp_out[i00] = temp_in[i00] + fact * (d2tdx2 + d2tdy2);
    }
  }
}

void handle_error(hipError_t err)
{
  if (err != hipSuccess)
  {
    fprintf(stderr, "GPUassert: %s\n", hipGetErrorString(err));
    exit(err);
  }
}

int main()
{
  int istep;
  int nstep = 200; // number of time steps

  float time;
  hipEvent_t start, stop;

  // Specify our 2D dimensions
  const int ni = NI;
  const int nj = NJ;
  float tfac = 8.418e-5; // thermal diffusivity of silver

  float *temp1_ref, *temp2_ref, *temp1, *temp2, *temp_tmp, *cpu_arr;

  const size_t size = ni * nj * sizeof(float);

  temp1_ref = (float *)malloc(size);
  temp2_ref = (float *)malloc(size);
  cpu_arr = (float *)malloc(size);

  handle_error(hipMalloc((void **)&temp1, size));

  handle_error(hipMalloc((void **)&temp2, size));

  // Initialize with random data
  for (int i = 0; i < ni * nj; ++i)
  {
    float rnd = (float)rand() / (float)(RAND_MAX / 100.0f);
    temp1_ref[i] = temp2_ref[i] = rnd;
  }

  handle_error(hipMemcpy(temp1, temp1_ref, size, hipMemcpyHostToDevice));
  handle_error(hipMemcpy(temp2, temp2_ref, size, hipMemcpyHostToDevice));

  // Execute the CPU-only reference version
  handle_error(hipEventCreate(&start));
  handle_error(hipEventCreate(&stop));
  handle_error(hipEventRecord(start, 0));
  for (istep = 0; istep < nstep; istep++)
  {
    step_kernel_ref(ni, nj, tfac, temp1_ref, temp2_ref);

    // swap the temperature pointers
    temp_tmp = temp1_ref;
    temp1_ref = temp2_ref;
    temp2_ref = temp_tmp;
  }
  handle_error(hipEventRecord(stop, 0));
  handle_error(hipEventSynchronize(stop));
  handle_error(hipEventElapsedTime(&time, start, stop));
  printf("Time CPU: %3.1f ms \n", time);

  // Execute the modified version using same data
  // https://developer.nvidia.com/blog/cuda-refresher-cuda-programming-model/
  dim3 threadsPerBlock(THREADS_X, THREADS_Y); // 1024 threads per block
  dim3 numBlocks((ni + threadsPerBlock.x - 1) / threadsPerBlock.x, (nj + threadsPerBlock.y - 1) / threadsPerBlock.y);

  handle_error(hipEventCreate(&start));
  handle_error(hipEventCreate(&stop));
  handle_error(hipEventRecord(start, 0));
  for (istep = 0; istep < nstep; istep++)
  {
    step_kernel_mod<<<numBlocks, threadsPerBlock>>>(ni, nj, tfac, temp1, temp2);

    // swap the temperature pointers
    temp_tmp = temp1;
    temp1 = temp2;
    temp2 = temp_tmp;
  }
  handle_error(hipEventRecord(stop, 0));
  handle_error(hipEventSynchronize(stop));
  handle_error(hipEventElapsedTime(&time, start, stop));
  printf("Time GPU: %3.1f ms \n", time);

  handle_error(hipMemcpy(cpu_arr, temp1, size, hipMemcpyDeviceToHost));

  float maxError = 0;
  // Output should always be stored in the temp1 and temp1_ref at this point
  for (int i = 0; i < ni * nj; ++i)
  {
    if (abs(cpu_arr[i] - temp1_ref[i]) > maxError)
    {
      // printf("cpu_arr: %f - temp1_ref:%f\n",cpu_arr[i] ,temp1_ref[i] );
      maxError = abs(cpu_arr[i] - temp1_ref[i]);
    }
  }

  // Check and see if our maxError is greater than an error bound
  if (maxError > 0.0005f)
  {
    printf("Problem! The Max Error of %.5f is NOT within acceptable bounds.\n", maxError);
  }
  else
  {
    printf("The Max Error of %.5f is within acceptable bounds.\n", maxError);
  }

  free(temp1_ref);
  free(temp2_ref);
  handle_error(hipFree(temp1));
  handle_error(hipFree(temp2));

  return 0;
}
