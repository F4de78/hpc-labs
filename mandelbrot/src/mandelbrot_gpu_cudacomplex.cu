
#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>
//#include <complex>
#include <chrono>
#include <hip/hip_complex.h>

#define FTYPE double
#if FTYPE == double
    typedef double __ftype;
#else
    typedef float __ftype;
#endif

#ifndef THREADS_X
    #define THREADS_X 16
#endif
#ifndef THREADS_Y
    #define THREADS_Y 16
#endif

// Ranges of the set
#define MIN_X -2
#define MAX_X 1
#define MIN_Y -1
#define MAX_Y 1

// Image ratio
#define RATIO_X (MAX_X - MIN_X)
#define RATIO_Y (MAX_Y - MIN_Y)

// Image size
#ifndef RESOLUTION
    #define RESOLUTION 1000
#endif
#define WIDTH (RATIO_X * RESOLUTION)
#define HEIGHT (RATIO_Y * RESOLUTION)

#define STEP ((__ftype)RATIO_X / WIDTH)

#ifndef DEGREE
    #define DEGREE 2        // Degree of the polynomial
#endif

#ifndef ITERATIONS
    #define ITERATIONS 1000 // Maximum number of iterations
#endif
using namespace std;

__global__ void mandelbrot(int *const image) {

    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;

    if (col >= WIDTH || row >= HEIGHT) {
        return;
    }
    
    int pos = row * WIDTH + col;
    
    __ftype c_re = col * STEP + MIN_X;
    __ftype c_im = row * STEP + MIN_Y;

    hipDoubleComplex c = make_hipDoubleComplex(c_re, c_im);
    hipDoubleComplex z = make_hipDoubleComplex(0, 0);

    // z = z^2 + c

    for (int i = 1; i <= ITERATIONS; i++)
    {
        hipDoubleComplex result = hipCmul(z, z);
        z = hipCadd(result, c);

        // If it is convergent
        if (hipCabs(z) >= 2)
        {
            image[pos] = i;
            return;
        }
    }
}

void handle_error(hipError_t err){
    if(err != hipSuccess){
        cerr << "GPUassert: " << hipGetErrorString(err) << endl;
        exit(err);
    }
}

int main(int argc, char **argv)
{   
    cout << "Width: " << WIDTH << ", Height: " << HEIGHT << ", Step: "<< STEP << endl;
    int *const image = new int[HEIGHT * WIDTH];
    const int size = HEIGHT * WIDTH * sizeof(int);
    
    memset(image, 0, sizeof(int)*HEIGHT*WIDTH);

    int *image_gpu = nullptr;

    const auto start = chrono::steady_clock::now();

    handle_error(hipMalloc((void **)&image_gpu, size));

    hipMemcpy(image_gpu, image, size, hipMemcpyHostToDevice);

    dim3 threadsPerBlock(THREADS_X, THREADS_Y);
    dim3 numBlocks((WIDTH + threadsPerBlock.x - 1) / threadsPerBlock.x, (HEIGHT + threadsPerBlock.y - 1) / threadsPerBlock.y);
   
    mandelbrot<<<numBlocks, threadsPerBlock>>>(image_gpu);

    handle_error(hipMemcpy(image, image_gpu, size, hipMemcpyDeviceToHost));    
    
    const auto end = chrono::steady_clock::now();
    cout << "Time elapsed: "
         << chrono::duration_cast<chrono::milliseconds>(end - start).count()
         << " ms." << endl;

    // Write the result to a file
    ofstream matrix_out;

    if (argc < 2)
    {
        cout << "Please specify the output file as a parameter." << endl;
        return -1;
    }

    matrix_out.open(argv[1], ios::trunc);
    if (!matrix_out.is_open())
    {
        cout << "Unable to open file." << endl;
        return -2;
    }

    for (int row = 0; row < HEIGHT; row++)
    {
        for (int col = 0; col < WIDTH; col++)
        {
            matrix_out << image[row * WIDTH + col];

            if (col < WIDTH - 1)
                matrix_out << ',';
        }
        if (row < HEIGHT - 1)
            matrix_out << endl;
    }
    matrix_out.close();

    handle_error(hipFree(image_gpu));
    delete[] image; // It's here for coding style, but useless
    return 0;
}